#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib> // For rand()
#include <ctime>   // For seeding rand()

hipError_t multiplyWithCuda(float* c, const float* a, const float* b, unsigned int size);

void printMatrix(const float* matrix, int rows, int columns, const char* name) {
    printf("Matrix %s:\n", name);
    for (int i = 0; i < rows; ++i) {
        printf("{ "); // Start of row delimiter
        for (int j = 0; j < columns; ++j) {
            printf("%.2f", matrix[i * columns + j]); // Print with 2 decimal place
            if (j < columns - 1) {
                printf(" "); // Add space between elements
            }
        }
        printf(" }"); // End of row delimiter
        printf("\n"); // Newline after each row
    }
    printf("\n");
}



float* createArray(int rows, int columns) {
    int totalSize = rows * columns;

    float* array = new float[totalSize];

    for (int i = 0; i < totalSize; ++i) {
        array[i] = static_cast<float>(std::rand()) / RAND_MAX * 4.0f; // Random floats [0, 4]    
    }
    return array;
}


__global__ void multiplyKernel(float* c, const float* a, const float* b)
{
    int row = (blockIdx.y * blockDim.y) + threadIdx.y;
    int column = (blockIdx.x * blockDim.x) + threadIdx.x;
    int matC = (row * 8 + column);
        
    int linearIndexRow;
    int linearIndexColumn;

    float sum=0;
    //Cette condition permet de rester dans les limites 
    if (row < 8){
        for (int index = 0; index < 8; index++) {
            linearIndexRow = (row * 8) + index;
            linearIndexColumn = (index*8)+column;
            sum += a[linearIndexRow] * b[linearIndexColumn];
        }
        printf("%.2f", sum);

        c[matC] = sum;
    }
    
}


int main()
{
    const int arraySize = 8;
    const float* a = createArray(arraySize, arraySize);
    const float* b = createArray(arraySize, arraySize);
    float c[(arraySize*arraySize)] = { 0 };

    // Print matrices A and B before computation
    printMatrix(a, arraySize, arraySize, "A");
    printMatrix(b, arraySize, arraySize, "B");

    // Add vectors in parallel.
    hipError_t cudaStatus = multiplyWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // Print result matrix C after computation
    printMatrix(c, arraySize, arraySize, "C");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t multiplyWithCuda(float* c, const float* a, const float* b, unsigned int size)
{
    float* dev_a = 0;
    float* dev_b = 0;
    float* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, (size * size) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, (size * size) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, (size * size) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, (size * size) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    cudaStatus = hipMemcpy(dev_b, b, (size * size) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
        

    // Launch a kernel on the GPU with one thread for each element.
    dim3 threadsParBlock(8, 8, 1);
    dim3 nombreDeBlock((size/8), (size/8), 1);
    multiplyKernel << <nombreDeBlock, threadsParBlock >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, (size * size) * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
