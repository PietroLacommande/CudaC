#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>

#include <stdio.h>
#include <cstdlib> // For rand()
#include <ctime>   // For seeding rand()

#define tileLength (int)2


hipError_t multiplyWithCuda(float* c, const float* a, const float* b, unsigned int size);

void printMatrix(const float* matrix, int rows, int columns, const char* name) {
    printf("Matrix %s:\n", name);
    for (int i = 0; i < rows; ++i) {
        printf("{ "); // Start of row delimiter
        for (int j = 0; j < columns; ++j) {
            printf("%.2f", matrix[i * columns + j]); // Print with 2 decimal place
            if (j < columns - 1) {
                printf(" "); // Add space between elements
            }
        }
        printf(" }"); // End of row delimiter
        printf("\n"); // Newline after each row
    }
    printf("\n");
}



float* createArray(int rows, int columns) {
    int totalSize = rows * columns;

    float* array = new float[totalSize];

    for (int i = 0; i < totalSize; ++i) {
        array[i] = static_cast<float>(std::rand()) / RAND_MAX * 4.0f; // Random floats [0, 4]    
    }
    return array;
}


__global__ void multiplyKernel(float* c, const float* a, const float* b, int length) {
    int threadY = threadIdx.y;
    int threadX = threadIdx.x;

    int row = (tileLength * blockIdx.y) + threadY;
    int column = (tileLength * blockIdx.x) + threadX;

    __shared__ float tileA[tileLength][tileLength]; 
    __shared__ float tileB[tileLength][tileLength];

    float sum = 0;

    int numTiles = (length + tileLength - 1) / tileLength; 

    for (int t = 0; t < numTiles; t++) {
        // Data loading
        if (row < length && (t * tileLength + threadX) < length) {
            tileA[threadY][threadX] = a[row * length + (t * tileLength + threadX)];
        }
        else {
            tileA[threadY][threadX] = 0.0f;
        }

        if (column < length && (t * tileLength + threadY) < length) {
            tileB[threadY][threadX] = b[(t * tileLength + threadY) * length + column];
        }
        else {
            tileB[threadY][threadX] = 0.0f;
        }

        __syncthreads();

        for (int k = 0; k < tileLength; k++) {
            sum += tileA[threadY][k] * tileB[k][threadX];
        }

        __syncthreads();
    }

    if (row < length && column < length) {  
        c[(row * length) + column] = sum;
    }
}



int main()
{
    const int matrixSize = 16;
    const float* a = createArray(matrixSize, matrixSize);
    const float* b = createArray(matrixSize, matrixSize);
    float c[(matrixSize * matrixSize)] = { 0 };

    // Print matrices A and B before computation
    printMatrix(a, matrixSize, matrixSize, "A");
    printMatrix(b, matrixSize, matrixSize, "B");

    // Add vectors in parallel.
    hipError_t cudaStatus = multiplyWithCuda(c, a, b, matrixSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // Print result matrix C after computation
    printMatrix(c, matrixSize, matrixSize, "C");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t multiplyWithCuda(float* c, const float* a, const float* b, unsigned int size)
{
    float* dev_a = 0;
    float* dev_b = 0;
    float* dev_c = 0;
    hipError_t cudaStatus;

    if (size >64) {
        printf("Taille de la matrice limité à 64 par 64");
        return hipErrorInvalidValue;
    }

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, (size * size) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, (size * size) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, (size * size) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, (size * size) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    cudaStatus = hipMemcpy(dev_b, b, (size * size) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



    dim3 threadsParBlock(tileLength, tileLength, 1);
    dim3 nombreDeBlock((size + tileLength - 1) / tileLength, (size + tileLength - 1) / tileLength);

    multiplyKernel << <nombreDeBlock, threadsParBlock >> > (dev_c, dev_a, dev_b, size);


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, (size * size) * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
