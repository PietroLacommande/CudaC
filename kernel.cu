#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib> // For rand()
#include <ctime>   // For seeding rand()

hipError_t addWithCuda(float *c, const float *a, const float *b, unsigned int size);

void printMatrix(const float* matrix, int rows, int columns, const char* name) {
    printf("Matrix %s:\n", name);
    for (int i = 0; i < rows; ++i) {
        printf("{ "); // Start of row delimiter
        for (int j = 0; j < columns; ++j) {
            printf("%.2f", matrix[i * columns + j]); // Print with 1 decimal place
            if (j < columns - 1) {
                printf(" "); // Add space between elements
            }
        }
        printf(" }"); // End of row delimiter
        printf("\n"); // Newline after each row
    }
    printf("\n");
}



float* createArray(int rows, int columns) {
    int totalSize = rows * columns;

    float* array = new float[totalSize];

    for (int i = 0; i < totalSize; ++i) {
        array[i] = static_cast<float>(std::rand()) / RAND_MAX * 4.0f; // Random floats [0, 4]    
    }
    return array;
}

__global__ void addKernel(float *c, const float *a, const float *b)
{
    int row = (blockIdx.y * blockDim.y) + threadIdx.y;
    int column = (blockIdx.x * blockDim.x) + threadIdx.x;

    
    int linearIndex = (row * 64) + column;
    
    //Cette condition permet de rester dans les limites 
    if (row < 64 && column < 64) {
        c[linearIndex] = a[linearIndex] + b[linearIndex];
    }
    
}

int main()
{
    const int arraySize = 4096;
    const float* a = createArray(64, 64);
    const float* b = createArray(64, 64);
    float c[arraySize] = { 0 };

    // Print matrices A and B before computation
    printMatrix(a, 64, 64, "A");
    printMatrix(b, 64, 64, "B");

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // Print result matrix C after computation
    printMatrix(c, 64, 64, "C");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float *c, const float *a, const float *b, unsigned int size)
{
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    // Cette config de grid, blocks, threads représente parfaitement une matrice de 64x64 tout en ayant un nombre de threads par block optimal
    dim3 threadsParBlock(16, 16, 1);
    dim3 nombreDeBlock(4, 4, 1);
    addKernel<<<nombreDeBlock, threadsParBlock >>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
