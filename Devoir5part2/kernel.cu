#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib> // For rand()
#include <ctime>   // For seeding rand()

hipError_t multiplyWithCuda(float* a, const float* b, const float* c, unsigned int size);

void printMatrix(const float* matrix, int rows, int columns, const char* name) {
    printf("Matrix %s:\n", name);
    for (int i = 0; i < rows; ++i) {
        printf("{ "); // Start of row delimiter
        for (int j = 0; j < columns; ++j) {
            printf("%.2f", matrix[i * columns + j]); // Print with 2 decimal place
            if (j < columns - 1) {
                printf(" "); // Add space between elements
            }
        }
        printf(" }"); // End of row delimiter
        printf("\n"); // Newline after each row
    }
    printf("\n");
}



float* createArray(int rows, int columns) {
    int totalSize = rows * columns;

    float* array = new float[totalSize];

    for (int i = 0; i < totalSize; ++i) {
        array[i] = static_cast<float>(std::rand()) / RAND_MAX * 4.0f; // Random floats [0, 4]    
    }
    return array;
}


__global__ void multiplyKernel(float* a, const float* b, const float* c)
{
    int row = (blockIdx.y * blockDim.y) + threadIdx.y;
    int linearIndex;
    float sum = 0.0;
    

    for (int index=0; index < 64 && row <64; index++) {
        linearIndex = (row * 64) + index;
        sum += b[linearIndex] * c[index];
    }
    a[row] = sum;
}


int main()
{
    const int arraySize = 64;
    const float* b = createArray(64, 64);
    const float* c = createArray(64, 1);
    float a[arraySize] = { 0 };

    // Print matrices A and B before computation
    printMatrix(b, 64, 64, "B");
    printMatrix(c, 64, 1, "C");

    // Add vectors in parallel.
    hipError_t cudaStatus = multiplyWithCuda(a, b, c, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // Print result matrix C after computation
    printMatrix(a, 64, 1, "A");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t multiplyWithCuda(float* a, const float* b, const float* c, unsigned int size)
{
    float* dev_a = 0;
    float* dev_b = 0;
    float* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, (size*size) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_b, b, (size*size)* sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_c, c, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    // Cette config de grid, blocks, threads représente parfaitement une matrice de 64x64 tout en ayant un nombre de threads par block optimal
    dim3 threadsParBlock(16, 16, 1);
    dim3 nombreDeBlock(4, 4, 1);
    multiplyKernel << <nombreDeBlock, threadsParBlock >> > (dev_a, dev_b, dev_c);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(a, dev_a, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
